#include "hip/hip_runtime.h"
/** TRACCC library, part of the ACTS project (R&D line)
 *
 * (c) 2022 CERN for the benefit of the ACTS project
 *
 * Mozilla Public License Version 2.0
 */

// CUDA Library include(s).
#include "traccc/cuda/clusterization/clusterization_algorithm.hpp"

// Project include(s)
#include "traccc/clusterization/device/connect_components.hpp"
#include "traccc/clusterization/device/count_cluster_cells.hpp"
#include "traccc/clusterization/device/create_measurements.hpp"
#include "traccc/clusterization/device/find_clusters.hpp"
#include "traccc/clusterization/device/form_spacepoints.hpp"
#include "traccc/cuda/utils/make_prefix_sum_buff.hpp"
#include "traccc/device/fill_prefix_sum.hpp"

// Vecmem include(s).
#include <vecmem/utils/copy.hpp>
#include <vecmem/utils/cuda/copy.hpp>

// System include(s).
#include <algorithm>
#include <string>
#include <chrono>

// Local include(s)
#include "traccc/cuda/utils/definitions.hpp"

namespace traccc::cuda {
namespace kernels {


// __global__ void setup_cluster_labels(
//     vecmem::data::vector_view<std::size_t> cell_to_module_view,
//     vecmem::data::vector_view<std::size_t> cell_indices_in_mod_view,
//     vecmem::data::jagged_vector_view<unsigned int> cell_cluster_label_view) {
//     /*
//         * This function sets up labels for every cell. Before running
//         * Hoshen-Kopelman, each vector in the jagged vector cell_cluster_label_view
//         * (which corresponds to one module each), is changed from being all
//         * zeros to an array from 1 to N, where N is the number of cells in
//         * the given module that are activated. Note that because we are dealing
//         * with an inconsistent device vector, we just launch K threads, where K
//         * is the total number of cells. This is in place of doing a N threads
//         * for each module, the (module, cell) number being represented by (x,y)
//         * in the threadIdx.
//         */
//     // get global index
//     unsigned int global_idx = threadIdx.x + blockIdx.x * blockDim.x;

//     device::set_init_cluster_labels(global_idx, cell_to_module_view, 
//                                     cell_indices_in_mod_view,
//                                     cell_cluster_label_view);
// }


__global__ void find_clusters_cell_parallel(
    const cell_container_types::const_view cells_view,
    vecmem::data::vector_view<std::size_t> cell_to_module_view,
    vecmem::data::vector_view<std::size_t> cell_indices_in_mod_view,
    vecmem::data::jagged_vector_view<unsigned int> cell_cluster_label_view,
    vecmem::data::vector_view<std::size_t> clusters_per_module_view) {
    /*
     * this function is the same as find_clusters but instead of every module
     * being a thread, every cell is a thread instead. Thus, it has an
     * extra argument which makes it possible to map the current cell (idx) to
     * the module it belongs to.
     */
    unsigned int thread_idx = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Initialize the device container for cells
    cell_container_types::const_device cells_device(cells_view);
    // Do the same with cell to module and cell index mapping
    vecmem::device_vector<std::size_t> device_cell_to_module(
        cell_to_module_view);
    vecmem::device_vector<std::size_t> device_cell_indices_in_mod(
        cell_indices_in_mod_view);

    // Ignore if idx is out of range
    if (thread_idx >= device_cell_to_module.size())
        return;

    // get the current module number from the current cell idx
    std::size_t module_number = device_cell_to_module.at(thread_idx);
    std::size_t cell_index = device_cell_indices_in_mod.at(thread_idx);

    // Initialise the jagged device vector for cell cluster indices
    // and the device vector for the number of clusters per module
    vecmem::jagged_device_vector<unsigned int> device_cell_cluster_labels(
        cell_cluster_label_view);
    vecmem::device_vector<std::size_t> device_clusters_per_module(
        clusters_per_module_view);

    // Get the cells for the current module and the cell this thread
    // is looking at
    const vecmem::device_vector<const traccc::cell>& cells =
        cells_device.at(module_number).items;
    // Get the relevant labels, so the ones for this current module
    vecmem::device_vector<unsigned int> cluster_labels = 
        device_cell_cluster_labels[module_number];
    
    // bool label_changed = true;
    // unsigned int neighbour_index = 0;
    // neighbour_index--;  // wrap around to be max to ensure no init neighbour def
    // unsigned int n_iters = 0;

    // while(label_changed) {
    //     label_changed = device::find_clusters_cell_parallel_passthrough(
    //                     module_number, cell_index, cells,
    //                     cluster_labels, neighbour_index);
    //     __syncthreads();
    //     n_iters++;
    // }
    // // if (n_iters >= 10) {
    // //     printf("%d iterations in cell %d.\n", n_iters, cell_idx);
    // // }
    unsigned int NN_index =
        device::setup_cluster_labels_and_NN(cell_index, cells, cluster_labels);

    if (NN_index == cells.size()) {
        //printf("Found an origin cell!\n");
        // we have hit an origin label, set it
        std::size_t* cluster_size = &device_clusters_per_module[module_number];
        unsigned int* cluster_size_uint = (unsigned int*) cluster_size;
        unsigned int cluster_label = atomicAdd(cluster_size_uint, 1) + 1;
        // device::write_value(&cluster_labels[cell_index], cluster_label);
        cluster_labels[cell_index] = cluster_label;
    }

    // first write NN value into current
    //device::write_from_NN(cell_index, cells, cluster_labels);
    // ensure all cells have found their NN before continuing:
    __syncthreads();
    // lastly, look through the labels and assign iteratively
    device::hk_find(cell_index, cluster_labels);
}


__global__ void normalise_cluster_numbers(
    vecmem::data::jagged_vector_view<unsigned int> cell_cluster_label_view,
    vecmem::data::vector_view<std::size_t> clusters_per_module_view) {

    /*
     * This function sets the cluster labels to be 1 -> N, where N is the
     * number of clusters in the module. After the initial HK clusterisation,
     * there are still N unique labels only, but they are not necessarily
     * numbered 1 -> N. Furthermore, this function also writes the number of
     * labels in a module (so N) to the clusters_per_module vector.
     */
    unsigned int module_number = threadIdx.x + blockIdx.x * blockDim.x;
    device::normalise_cluster_numbers(module_number, cell_cluster_label_view,
                                      clusters_per_module_view);
}


__global__ void find_clusters(
    const cell_container_types::const_view cells_view,
    vecmem::data::jagged_vector_view<unsigned int> cell_cluster_label_view,
    vecmem::data::vector_view<std::size_t> clusters_per_module_view) {

    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    device::find_clusters(idx, cells_view, cell_cluster_label_view,
                          clusters_per_module_view);
}


__global__ void print_debug_module(
    const cell_container_types::const_view cells_view,
    vecmem::data::jagged_vector_view<unsigned int> cell_cluster_label_view,
    vecmem::data::vector_view<std::size_t> clusters_per_module_view,
    bool check_all, bool before_clust, unsigned int module_to_check) {
    /*
     * This is a debug function which prints information about the current
     * module. Current module is the global threadIdx.
     */

    cell_container_types::const_device cells_device(cells_view);
    vecmem::jagged_device_vector<unsigned int> device_cell_cluster_label(
        cell_cluster_label_view);
    vecmem::device_vector<std::size_t> device_clusters_per_module(
        clusters_per_module_view);

    // 755 has the most cells
    // 2464 has exactly one cluster with the most cells (5)
    // module_to_check = 755;
    
    int module_number = threadIdx.x + blockIdx.x * blockDim.x;
    // if all modules are to be checked, then check if index is in range
    // otherwise, check only if index is right
    bool check = (check_all && (module_number < cells_device.size())) ||
                 (module_number == module_to_check);
    if (check && before_clust) {
        // first print the cells for this module
        const vecmem::device_vector<const traccc::cell>& cells =
            cells_device.at(module_number).items;
        const traccc::cell_module cells_header =
            cells_device.at(module_number).header;
        size_t n_cells = cells.size();

        // TODO figure out what "module" means here
        printf("cells_header: id: %d, module: %d range0: (%d, %d), range1: (%d, %d)\n",
               (int) cells_header.event, (int) cells_header.module, 
               (int) cells_header.range0[0], (int) cells_header.range0[1],
               (int) cells_header.range1[0], (int) cells_header.range1[1]);

        for (int i=0; i < n_cells; i++) {
            const traccc::cell cell = cells[i];
            scalar act = cell.activation;
            printf("Idx: %d, i.e. (%d, %d): Cell index %d and position: (%d, %d) activation: %f\n",
                    module_number, blockIdx.x, threadIdx.x, i, cell.channel0, cell.channel1, act);

        }
        // then print the inputs/outputs that go into the actual algorithm

        // THE BELOW IS COMMENTED BECAUSE IT'S NOT REALLY NECESSARY, ALL CELLS ARE ALLOCATED
        // TO CLUSTER NUMBER 0 AT INITIALISATION

        // unsigned int n_cell_cluster_label = device_cell_cluster_label.at(idx).size();
        // for (int i=0; i < n_cell_cluster_label; i++) {
        //     unsigned int cell_cluster_number = device_cell_cluster_label.at(idx).at(i);
        //     printf("Before: Idx: %d, Clusters in module: %d, Cell %d belongs to cluster %d\n",
        //         idx, (int) device_clusters_per_module.at(idx), i, cell_cluster_number);
        // }

        printf("Before: Idx: %d, Clusters in module: %d. All cells are in cluster 0.\n",
            module_number, (int) device_clusters_per_module.at(module_number));
    }

    if (check && !before_clust) {
        const vecmem::device_vector<const traccc::cell>& cells =
            cells_device.at(module_number).items;
        unsigned int n_cell_cluster_label =
            device_cell_cluster_label.at(module_number).size();
        //printf("n_cell_cluster_label: %u\n", n_cell_cluster_label);
        // print outputs from clusterisation algo
        // printf("After: %d clusters in module %d.\n",
        //        (int) device_clusters_per_module.at(module_number), (int) module_number);

        for (int i=0; i < n_cell_cluster_label; i++) {
            const traccc::cell cell = cells[i];
            unsigned int cell_cluster_number =
                device_cell_cluster_label.at(module_number).at(i);
            printf("After: Module: %d, Clusters in module: %d, Cell %d, Point (%d, %d) belongs to cluster %d. Activation: %f\n",
                module_number, (int) device_clusters_per_module.at(module_number), i, cell.channel0,
                cell.channel1, cell_cluster_number, cell.activation);
        }
    }
    // if (device_clusters_per_module.at(idx) == 2) {
    //     printf("Module with two clusters: %d. Number of cells: %d\n",
    //            idx, cells_device.at(idx).items.size());
    // }
    //printf("Module %d has %d clusters\n.", idx, device_clusters_per_module.at(idx));
}

__global__ void count_cluster_cells(
    vecmem::data::jagged_vector_view<unsigned int> cell_cluster_label_view,
    vecmem::data::vector_view<std::size_t> cluster_prefix_sum_view,
    vecmem::data::vector_view<const device::prefix_sum_element_t>
        cells_prefix_sum_view,
    vecmem::data::vector_view<unsigned int> cluster_sizes_view) {

    device::count_cluster_cells(
        threadIdx.x + blockIdx.x * blockDim.x, cell_cluster_label_view,
        cluster_prefix_sum_view, cells_prefix_sum_view, cluster_sizes_view);
}

__global__ void connect_components(
    const cell_container_types::const_view cells_view,
    vecmem::data::jagged_vector_view<unsigned int> cell_cluster_label_view,
    vecmem::data::vector_view<std::size_t> cluster_prefix_sum_view,
    vecmem::data::vector_view<const device::prefix_sum_element_t>
        cells_prefix_sum_view,
    cluster_container_types::view clusters_view) {

    device::connect_components(threadIdx.x + blockIdx.x * blockDim.x,
                               cells_view, cell_cluster_label_view,
                               cluster_prefix_sum_view, cells_prefix_sum_view,
                               clusters_view);
}
__global__ void create_measurements(
    const cell_container_types::const_view cells_view,
    cluster_container_types::const_view clusters_view,
    measurement_container_types::view measurements_view) {

    device::create_measurements(threadIdx.x + blockIdx.x * blockDim.x,
                                clusters_view, cells_view, measurements_view);
}

__global__ void form_spacepoints(
    measurement_container_types::const_view measurements_view,
    vecmem::data::vector_view<const device::prefix_sum_element_t>
        measurements_prefix_sum_view,
    spacepoint_container_types::view spacepoints_view) {

    device::form_spacepoints(threadIdx.x + blockIdx.x * blockDim.x,
                             measurements_view, measurements_prefix_sum_view,
                             spacepoints_view);
}

}  // namespace kernels

clusterization_algorithm::clusterization_algorithm(
    const traccc::memory_resource& mr)
    : m_mr(mr) {

    // Initialize m_copy ptr based on memory resources that were given
    if (mr.host) {
        m_copy = std::make_unique<vecmem::cuda::copy>();
    } else {
        m_copy = std::make_unique<vecmem::copy>();
    }
}

clusterization_algorithm::output_type clusterization_algorithm::operator()(
    const cell_container_types::host& cells_per_event) const {

    // Vecmem copy object for moving the data between host and device
    vecmem::copy copy;

    printf("Starting CUDA clusterization.\n");
    // Number of modules
    unsigned int num_modules = cells_per_event.size();
    printf("Number of modules: %d\n", num_modules);

    // Work block size for kernel execution
    std::size_t threadsPerBlock = 64;
    std::size_t blocksPerGrid;  // initialise, will change dep. on kernel
    // for debug: Choose which level of parallelisation
    bool parallelise_by_cell = true;

    // Get the view of the cells container
    auto cells_data =
        get_data(cells_per_event, (m_mr.host ? m_mr.host : &(m_mr.main)));

    //int test = (int) cells_data;
    // auto *headers = &cells_data.headers;
    // auto *cells = vecmem::get_data(&cells_data.items);
    // for (unsigned int j = 0; j < num_modules; j++){
    //     auto module_cells = cells[j];
    //     std::sort(module_cells->begin(), module_cells->end(),
    //         [](auto a, auto b)
    //             {
    //                 return a.channel1 > b.channel1;
    //             });
    // }

    // Get the sizes of the cells in each module
    auto cell_sizes = copy.get_sizes(cells_data.items);
    unsigned int n_cells_total = 0;
    for (unsigned int i = 0; i < cell_sizes.size(); i++) {
        n_cells_total += cell_sizes[i];
    }
    // create a vector which maps the cell index to current module
    vecmem::vector<std::size_t> cell_to_module(n_cells_total);
    vecmem::vector<std::size_t> cell_indices_in_module(n_cells_total);
    unsigned int curr_idx = 0;  // used to populate the above
    for (std::size_t i = 0; i < cell_sizes.size(); i++) {
        // if (i == 3000) {
        //         printf("Module 3000 initialise: \n\n");
        //     }
        for (std::size_t j = 0; j < cell_sizes[i]; j++) {
            cell_to_module[curr_idx] = i;  // i is the module number
            // j is the cell number in the module
            cell_indices_in_module[curr_idx] = j;
            // if (i == 3000) {
            //     printf("cell_indices_in_module[%d] = %d\n", (int) curr_idx, (int) j);
            // }
            curr_idx++;
        }
    }
    // instantiate vector buffers to hold the above data
    vecmem::data::vector_buffer<std::size_t> cell_to_module_buff(
        n_cells_total, m_mr.main);
    m_copy->setup(cell_to_module_buff);
    
    vecmem::data::vector_buffer<std::size_t> cell_indices_in_mod_buff(
        n_cells_total, m_mr.main);
    m_copy->setup(cell_indices_in_mod_buff);
    
    // move the vectors to the device and create vector views
    (*m_copy)(vecmem::get_data(cell_to_module), cell_to_module_buff,
        vecmem::copy::type::copy_type::host_to_device);
    (*m_copy)(vecmem::get_data(cell_indices_in_module), cell_indices_in_mod_buff,
        vecmem::copy::type::copy_type::host_to_device);

    // create and move the vectors on the device to vector views
    vecmem::data::vector_view<std::size_t> cell_to_module_view =
        cell_to_module_buff;
    vecmem::data::vector_view<std::size_t> cell_indices_in_mod_view =
        cell_indices_in_mod_buff;
    /*
     * Helper container for clusterisation calculations.
     * Each inner vector corresponds to 1 module.
     * The indices in a particular inner vector will be filled by the
     * relevant clusterisation algorithm (sparse CCL or Hoshen-Kopelman)
     * and will indicate to which cluster, a particular cell in the module
     * belongs to.
     */
    vecmem::data::jagged_vector_buffer<unsigned int> cell_cluster_label_buff(
        std::vector<std::size_t>(cell_sizes.begin(), cell_sizes.end()),
        m_mr.main, m_mr.host);
    m_copy->setup(cell_cluster_label_buff);
    // Create view to pass to kernel which sets up cluster labels
    vecmem::data::jagged_vector_view<unsigned int> cell_cluster_label_view =
        cell_cluster_label_buff;

    // if (parallelise_by_cell) {
    //     // only if parallelising the clusterisation cell by cell do we
    //     // give each cell a label before clusterisation
    //     blocksPerGrid = (n_cells_total + threadsPerBlock - 1) / threadsPerBlock;

    //     // Invoke find clusters that will call cluster finding kernel
    //     kernels::setup_cluster_labels<<<blocksPerGrid, threadsPerBlock>>>(
    //         cell_to_module_view, cell_indices_in_mod_view, cell_cluster_label_view);

    //     CUDA_ERROR_CHECK(hipGetLastError());
    //     CUDA_ERROR_CHECK(hipDeviceSynchronize());
    // }

    /*
     * cl_per_module_prefix_buff is a vector buffer with numbers of found
     * clusters in each module. Later it will be transformed into prefix sum
     * vector (hence the name). The logic is the following. After
     * cluster_finding_kernel, the buffer will contain cluster sizes e.i.
     *
     * cluster sizes: | 1 | 12 | 5 | 102 | 42 | ... - cl_per_module_prefix_buff
     * module index:  | 0 |  1 | 2 |  3  |  4 | ...
     *
     * Now, we copy those cluster sizes to the host and make a duplicate vector
     * of them. So, we are left with cl_per_module_prefix_host, and
     * clusters_per_module_host - which are the same. Now, we procede to
     * modifying the cl_per_module_prefix_host to actually resemble its name
     * i.e.
     *
     * We do std::inclusive_scan on it, which will result in a prefix sum
     * vector:
     *
     * cl_per_module_prefix_host: | 1 | 13 | 18 | 120 | 162 | ...
     *
     * Then, we copy this vector into the previous cl_per_module_prefix_buff.
     * In this way, we don't need to allocate the memory on the device twice.
     *
     * Now, the monotonic prefix sum buffer - cl_per_module_prefix_buff, will
     * allow us to insert the clusters at the correct position inside the
     * kernel. The remaining host vector - clusters_per_module_host, will be
     * needed to allocate memory for other buffers later in the code.
     */
    vecmem::data::vector_buffer<std::size_t> cl_per_module_prefix_buff(
        num_modules, m_mr.main);
    m_copy->setup(cl_per_module_prefix_buff);

    // Create views to pass to cluster finding kernel
    const cell_container_types::const_view cells_view(cells_data);
    vecmem::data::vector_view<std::size_t> cl_per_module_prefix_view =
        cl_per_module_prefix_buff;

    if (parallelise_by_cell) {
        // get the grid size for using all cells
        threadsPerBlock = 1024;
        blocksPerGrid = (n_cells_total + threadsPerBlock - 1) / threadsPerBlock;

        auto start_clusterisation_time = std::chrono::system_clock::now();
        // Run cell parallelised kernel to get clusters
        kernels::find_clusters_cell_parallel<<<blocksPerGrid, threadsPerBlock>>>(
            cells_view, cell_to_module_view, cell_indices_in_mod_view,
            cell_cluster_label_view, cl_per_module_prefix_view);

        CUDA_ERROR_CHECK(hipGetLastError());
        CUDA_ERROR_CHECK(hipDeviceSynchronize());

        auto end_clusterisation_time = std::chrono::system_clock::now();

        std::chrono::duration<double> clusterisation_time =
            end_clusterisation_time - start_clusterisation_time;
        
        printf("TIME TAKEN FOR HK CLUSTERISATION: %fs\n", clusterisation_time.count());

        // go back to module wide parallelisation
        threadsPerBlock = 64;
        // blocksPerGrid = (num_modules + threadsPerBlock - 1) / threadsPerBlock;
        // kernels::print_debug_module<<<blocksPerGrid, threadsPerBlock>>>(
        //     cells_view, cell_cluster_label_view, cl_per_module_prefix_view,
        //     true, false, 3000);  // check for module 755 after clusterisation
        
        // CUDA_ERROR_CHECK(hipGetLastError());
        // CUDA_ERROR_CHECK(hipDeviceSynchronize());

        // normalise the output to have labels from just 1 -> N
        // kernels::normalise_cluster_numbers<<<blocksPerGrid, threadsPerBlock>>>(
        //     cell_cluster_label_view, cl_per_module_prefix_view);

        // CUDA_ERROR_CHECK(hipGetLastError());
        // CUDA_ERROR_CHECK(hipDeviceSynchronize());
        
        // auto end_normalisation_time = std::chrono::system_clock::now();
        // std::chrono::duration<double> normalisation_time =
        //     end_normalisation_time - end_clusterisation_time;
        
        // printf("TIME TAKEN FOR LABEL NORMALISATION: %fs\n", normalisation_time.count());
        // kernels::print_debug_module<<<blocksPerGrid, threadsPerBlock>>>(
        //     cells_view, cell_cluster_label_view, cl_per_module_prefix_view,
        //     false, false, 970);  // check for module 755 after clusterisation
        
        // CUDA_ERROR_CHECK(hipGetLastError());
        // CUDA_ERROR_CHECK(hipDeviceSynchronize());
    }
    else {  // parallelise by module
        // Calculating grid size for cluster finding kernel
        blocksPerGrid = (num_modules + threadsPerBlock - 1) / threadsPerBlock;

        auto start_clusterisation_time = std::chrono::system_clock::now();
        // Invoke find clusters that will call cluster finding kernel
        kernels::find_clusters<<<blocksPerGrid, threadsPerBlock>>>(
            cells_view, cell_cluster_label_view, cl_per_module_prefix_view);

        CUDA_ERROR_CHECK(hipGetLastError());
        CUDA_ERROR_CHECK(hipDeviceSynchronize());

        auto end_clusterisation_time = std::chrono::system_clock::now();

        std::chrono::duration<double> clusterisation_time =
            end_clusterisation_time - start_clusterisation_time;
        
        printf("TIME TAKEN FOR CCL CLUSTERISATION: %fs\n", clusterisation_time.count());
        
        // kernels::print_debug_module<<<blocksPerGrid, threadsPerBlock>>>(
        //     cells_view, cell_cluster_label_view, cl_per_module_prefix_view,
        //     false, false, 755);  // check for module 755 after clusterisation
        
        CUDA_ERROR_CHECK(hipGetLastError());
        CUDA_ERROR_CHECK(hipDeviceSynchronize());
    }


    // Create prefix sum buffer
    vecmem::data::vector_buffer cells_prefix_sum_buff =
        make_prefix_sum_buff(cell_sizes, *m_copy, m_mr);

    // Copy the sizes of clusters per module to the host
    // and create a copy of "clusters per module" vector
    vecmem::vector<std::size_t> cl_per_module_prefix_host(
        m_mr.host ? m_mr.host : &(m_mr.main));
    (*m_copy)(cl_per_module_prefix_buff, cl_per_module_prefix_host,
              vecmem::copy::type::copy_type::device_to_host);
    std::vector<std::size_t> clusters_per_module_host(
        cl_per_module_prefix_host.begin(), cl_per_module_prefix_host.end());

    // Perform the inclusive scan operation
    std::inclusive_scan(cl_per_module_prefix_host.begin(),
                        cl_per_module_prefix_host.end(),
                        cl_per_module_prefix_host.begin());

    unsigned int total_clusters = cl_per_module_prefix_host.back();

    // Copy the prefix sum back to its device container
    (*m_copy)(vecmem::get_data(cl_per_module_prefix_host),
              cl_per_module_prefix_buff,
              vecmem::copy::type::copy_type::host_to_device);

    // Vector of the exact cluster sizes, will be filled in cluster counting
    vecmem::data::vector_buffer<unsigned int> cluster_sizes_buffer(
        total_clusters, m_mr.main);
    m_copy->setup(cluster_sizes_buffer);
    m_copy->memset(cluster_sizes_buffer, 0);

    // Create views to pass to cluster counting kernel
    vecmem::data::vector_view<const device::prefix_sum_element_t>
        cells_prefix_sum_view = cells_prefix_sum_buff;
    vecmem::data::vector_view<unsigned int> cluster_sizes_view =
        cluster_sizes_buffer;

    // Calclating grid size for cluster counting kernel (block size 64)
    blocksPerGrid =
        (cells_prefix_sum_view.size() + threadsPerBlock - 1) / threadsPerBlock;
    // Invoke cluster counting will call count cluster cells kernel
    kernels::count_cluster_cells<<<blocksPerGrid, threadsPerBlock>>>(
        cell_cluster_label_view, cl_per_module_prefix_view,
        cells_prefix_sum_view, cluster_sizes_view);
    // Check for kernel launch errors and Wait for the cluster_counting kernel
    // to finish
    CUDA_ERROR_CHECK(hipGetLastError());
    CUDA_ERROR_CHECK(hipDeviceSynchronize());

    // Copy cluster sizes back to the host
    std::vector<unsigned int> cluster_sizes;
    (*m_copy)(cluster_sizes_buffer, cluster_sizes,
              vecmem::copy::type::copy_type::device_to_host);

    // Cluster container buffer for the clusters and headers (cluster ids)
    cluster_container_types::buffer clusters_buffer{
        {total_clusters, m_mr.main},
        {std::vector<std::size_t>(total_clusters, 0),
         std::vector<std::size_t>(cluster_sizes.begin(), cluster_sizes.end()),
         m_mr.main, m_mr.host}};
    m_copy->setup(clusters_buffer.headers);
    m_copy->setup(clusters_buffer.items);

    // Create views to pass to component connection kernel
    cluster_container_types::view clusters_view = clusters_buffer;

    // Using previous block size and thread size (64)
    // Invoke connect components will call connect components kernel
    kernels::connect_components<<<blocksPerGrid, threadsPerBlock>>>(
        cells_view, cell_cluster_label_view, cl_per_module_prefix_view,
        cells_prefix_sum_view, clusters_view);
    CUDA_ERROR_CHECK(hipGetLastError());
    CUDA_ERROR_CHECK(hipDeviceSynchronize());

    // Resizable buffer for the measurements
    measurement_container_types::buffer measurements_buffer{
        {num_modules, m_mr.main},
        {std::vector<std::size_t>(num_modules, 0), clusters_per_module_host,
         m_mr.main, m_mr.host}};
    m_copy->setup(measurements_buffer.headers);
    m_copy->setup(measurements_buffer.items);

    // Spacepoint container buffer to fill inside the spacepoint formation
    // kernel
    spacepoint_container_types::buffer spacepoints_buffer{
        {num_modules, m_mr.main},
        {std::vector<std::size_t>(num_modules, 0), clusters_per_module_host,
         m_mr.main, m_mr.host}};
    m_copy->setup(spacepoints_buffer.headers);
    m_copy->setup(spacepoints_buffer.items);

    // Create views to pass to measurement creation kernel
    measurement_container_types::view measurements_view = measurements_buffer;

    // Calculating grid size for measurements creation kernel (block size 64)
    blocksPerGrid =
        (clusters_view.headers.size() - 1 + threadsPerBlock) / threadsPerBlock;

    // Invoke measurements creation will call create measurements kernel
    kernels::create_measurements<<<blocksPerGrid, threadsPerBlock>>>(
        cells_view, clusters_view, measurements_view);

    // Check for kernel launch errors and Wait here for the measurements
    // creation kernel to finish
    CUDA_ERROR_CHECK(hipGetLastError());
    CUDA_ERROR_CHECK(hipDeviceSynchronize());

    // Create prefix sum buffer
    vecmem::data::vector_buffer meas_prefix_sum_buff = make_prefix_sum_buff(
        m_copy->get_sizes(measurements_buffer.items), *m_copy, m_mr);

    // Create views to run spacepoint formation
    vecmem::data::vector_view<const device::prefix_sum_element_t>
        meas_prefix_sum_view = meas_prefix_sum_buff;
    spacepoint_container_types::view spacepoints_view = spacepoints_buffer;

    // Using the same grid size as before
    // Invoke spacepoint formation will call form_spacepoints kernel
    kernels::form_spacepoints<<<blocksPerGrid, threadsPerBlock>>>(
        measurements_view, meas_prefix_sum_view, spacepoints_view);
    // Check for kernel launch errors and Wait for the spacepoint formation
    // kernel to finish
    CUDA_ERROR_CHECK(hipGetLastError());
    CUDA_ERROR_CHECK(hipDeviceSynchronize());
    return spacepoints_buffer;
}

}  // namespace traccc::cuda